#include "hip/hip_runtime.h"
Skip to content
Why GitHub? 
Team
Enterprise
Explore 
Marketplace
Pricing 
Search

Sign in
Sign up
davidbp13
/
TDT4200_Assignment7
100
Code
Issues
Pull requests
Actions
Projects
Security
Insights
Join GitHub today
GitHub is home to over 50 million developers working together to host and review code, manage projects, and build software together.

TDT4200_Assignment7/main.cu
@davidbp13
davidbp13 Final CUDA code
Latest commit 7aea541 on Nov 8, 2019
 History
 1 contributor
582 lines (491 sloc)  21 KB
 
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
extern "C" {
    #include "libs/bitmap.h"
}

/* Divide the problem into blocks of BLOCKX x BLOCKY threads */
#define BLOCKY 32
#define BLOCKX 32

#define ERROR_EXIT -1

#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %s %d\n", hipGetErrorName(code), hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// Convolutional Filter Examples, each with dimension 3,
// gaussian filter with dimension 5
// If you apply another filter, remember not only to exchange
// the filter but also the filterFactor and the correct dimension.

int const sobelYFilter[] = {-1, -2, -1,
                             0,  0,  0,
                             1,  2,  1};
float const sobelYFilterFactor = (float) 1.0;

int const sobelXFilter[] = {-1, -0, -1,
                            -2,  0, -2,
                            -1,  0, -1 , 0};
float const sobelXFilterFactor = (float) 1.0;


int const laplacian1Filter[] = {  -1,  -4,  -1,
                                 -4,  20,  -4,
                                 -1,  -4,  -1};

float const laplacian1FilterFactor = (float) 1.0;

int const laplacian2Filter[] = { 0,  1,  0,
                                 1, -4,  1,
                                 0,  1,  0};
float const laplacian2FilterFactor = (float) 1.0;

int const laplacian3Filter[] = { -1,  -1,  -1,
                                  -1,   8,  -1,
                                  -1,  -1,  -1};
float const laplacian3FilterFactor = (float) 1.0;


//Bonus Filter:

int const gaussianFilter[] = { 1,  4,  6,  4, 1,
                               4, 16, 24, 16, 4,
                               6, 24, 36, 24, 6,
                               4, 16, 24, 16, 4,
                               1,  4,  6,  4, 1 };

float const gaussianFilterFactor = (float) 1.0 / 256.0;


// Apply convolutional filter on image data
void applyFilter(unsigned char **out, unsigned char **in, unsigned int width, unsigned int height, int *filter, unsigned int filterDim, float filterFactor) {
  unsigned int const filterCenter = (filterDim / 2);
  for (unsigned int y = 0; y < height; y++) {
    for (unsigned int x = 0; x < width; x++) {
      int aggregate = 0;
      for (unsigned int ky = 0; ky < filterDim; ky++) {
        int nky = filterDim - 1 - ky;
        for (unsigned int kx = 0; kx < filterDim; kx++) {
          int nkx = filterDim - 1 - kx;

          int yy = y + (ky - filterCenter);
          int xx = x + (kx - filterCenter);
          if (xx >= 0 && xx < (int) width && yy >=0 && yy < (int) height)
            aggregate += in[yy][xx] * filter[nky * filterDim + nkx];
        }
      }
      aggregate *= filterFactor;
      if (aggregate > 0) {
        out[y][x] = (aggregate > 255) ? 255 : aggregate;
      } else {
        out[y][x] = 0;
      }
    }
  }
}

/************************* GPU Kernel *************************/
__global__ void device_calculate(unsigned char *out, unsigned char *in, unsigned int width, unsigned int height, int *filter, unsigned int filterDim, float filterFactor) {
  //A single pixel is assigned to one tread in each of the blocks 
  int x = blockIdx.x *  blockDim.x + threadIdx.x;
  int y = blockIdx.y *  blockDim.y + threadIdx.y;
	
  //Make sure that only threads with a valid pixel compute
  if ( (x < width) && (y  < height) ){
    unsigned int const filterCenter = (filterDim / 2);
    int aggregate = 0;
    for (unsigned int ky = 0; ky < filterDim; ky++) {
      int nky = filterDim - 1 - ky;
      for (unsigned int kx = 0; kx < filterDim; kx++) {
        int nkx = filterDim - 1 - kx;

        int yy = y + (ky - filterCenter);
        int xx = x + (kx - filterCenter);
        if (xx >= 0 && xx < (int) width && yy >=0 && yy < (int) height)
          aggregate += in[xx + yy * width] * filter[nky * filterDim + nkx];
      }
    }
    aggregate *= filterFactor;
    if (aggregate > 0) {
      out[x + y * width] = (aggregate > 255) ? 255 : aggregate;
    } else {
      out[x + y * width] = 0;
    }
  }
}

__global__ void device_shareInput_calculate(unsigned char *out, unsigned char *in, unsigned int width, unsigned int height, int *filter, unsigned int filterDim, float filterFactor) {
  //A single pixel is assigned to one tread in each of the blocks 
  int x = blockIdx.x *  blockDim.x + threadIdx.x;
  int y = blockIdx.y *  blockDim.y + threadIdx.y;
  
  // Move pixels into shared memory
  extern __shared__ unsigned char in_shared[];
  in_shared[(threadIdx.x + threadIdx.y *  blockDim.x)] = in[x + y * width];

  __syncthreads();

  unsigned int const filterCenter = (filterDim / 2);
  	
  //Make sure that only threads with a valid pixel compute
  if ( (x < width) && (y  < height) ){
    // If the pixel needs pixels only from its block, use shared memory
    if ( !(threadIdx.x == 0 || threadIdx.x == (blockDim.x-1) || threadIdx.y == 0 || threadIdx.y == (blockDim.y-1)) ){  
      int aggregate = 0;
      for (unsigned int ky = 0; ky < filterDim; ky++) {
        int nky = filterDim - 1 - ky;
        for (unsigned int kx = 0; kx < filterDim; kx++) {
          int nkx = filterDim - 1 - kx;

          int yy = threadIdx.y + (ky - filterCenter);
          int xx = threadIdx.x + (kx - filterCenter);
          if (xx >= 0 && xx < blockDim.x && yy >=0 && yy < blockDim.y)
            aggregate += in_shared[(xx + yy *  blockDim.x)] * filter[nky * filterDim + nkx];
        }
      }
      aggregate *= filterFactor;
      if (aggregate > 0) {
        out[x + y * width] = (aggregate > 255) ? 255 : aggregate;
      } else {
        out[x + y * width] = 0;
      }
    }
    // If the pixel needs pixels from other blocks, use global memory 
    else{
      int aggregate = 0;
      for (unsigned int ky = 0; ky < filterDim; ky++) {
        int nky = filterDim - 1 - ky;
        for (unsigned int kx = 0; kx < filterDim; kx++) {
          int nkx = filterDim - 1 - kx;

          int yy = y + (ky - filterCenter);
          int xx = x + (kx - filterCenter);
          if (xx >= 0 && xx < (int) width && yy >=0 && yy < (int) height)
            aggregate += in[xx + yy *  width] * filter[nky * filterDim + nkx];
        }
      }
      aggregate *= filterFactor;
      if (aggregate > 0) {
        out[x + y * width] = (aggregate > 255) ? 255 : aggregate;
      } else {
        out[x + y * width] = 0;
      }
      
    }    
  }
}

__global__ void device_shareFilter_calculate(unsigned char *out, unsigned char *in, unsigned int width, unsigned int height, int *filter, unsigned int filterDim, float filterFactor) { 
  //A single pixel is assigned to one tread in each of the blocks 
  int x = blockIdx.x *  blockDim.x + threadIdx.x;
  int y = blockIdx.y *  blockDim.y + threadIdx.y;

  // Move filter into shared memory
  extern __shared__ int filter_shared[];
  
  /*
  for (int i = 0; i < filterDim*filterDim; i++){
    filter_shared[i] = filter[i];
  }*/
  if (threadIdx.x + threadIdx.y *  blockDim.x < filterDim * filterDim){
    filter_shared[threadIdx.x + threadIdx.y *  blockDim.x] = filter[threadIdx.x + threadIdx.y *  blockDim.x];
  }

  __syncthreads();

  unsigned int const filterCenter = (filterDim / 2);
  	
  //Make sure that only threads with a valid pixel compute
  if ( (x < width) && (y  < height) ){

      int aggregate = 0;
      for (unsigned int ky = 0; ky < filterDim; ky++) {
        int nky = filterDim - 1 - ky;
        for (unsigned int kx = 0; kx < filterDim; kx++) {
          int nkx = filterDim - 1 - kx;

          int yy = y + (ky - filterCenter);
          int xx = x + (kx - filterCenter);
          if (xx >= 0 && xx < (int) width && yy >=0 && yy < (int) height)
            aggregate += in[xx + yy *  width] * filter_shared[nky * filterDim + nkx];
        }
      }
      aggregate *= filterFactor;
      if (aggregate > 0) {
        out[x + y * width] = (aggregate > 255) ? 255 : aggregate;
      } else {
        out[x + y * width] = 0;
      }
          
  }
}

__global__ void device_shareInputandFilter_calculate(unsigned char *out, unsigned char *in, unsigned int width, unsigned int height, int *filter, unsigned int filterDim, float filterFactor) {
  //A single pixel is assigned to one tread in each of the blocks 
  int x = blockIdx.x *  blockDim.x + threadIdx.x;
  int y = blockIdx.y *  blockDim.y + threadIdx.y;
  
  extern __shared__ unsigned char shared_array[];
  
  // Move pixels into shared memory
  unsigned char* in_shared = (unsigned char*) shared_array;
  in_shared[threadIdx.x + threadIdx.y *  blockDim.x] = in[x + y * width];

  __syncthreads();

  // Move filter into shared memory
  int* filter_shared = (int*)&shared_array[blockDim.x*blockDim.y];
  
  if (threadIdx.x + threadIdx.y *  blockDim.x < filterDim * filterDim){
    filter_shared[threadIdx.x + threadIdx.y *  blockDim.x] = filter[threadIdx.x + threadIdx.y *  blockDim.x];
  }

  __syncthreads();

  unsigned int const filterCenter = (filterDim / 2);
  	
  //Make sure that only threads with a valid pixel compute
  if ( (x < width) && (y  < height) ){
    // If the pixel needs pixels only from its block, use shared memory
    if ( !(threadIdx.x == 0 || threadIdx.x == (blockDim.x-1) || threadIdx.y == 0 || threadIdx.y == (blockDim.y-1)) ){  
      int aggregate = 0;
      for (unsigned int ky = 0; ky < filterDim; ky++) {
        int nky = filterDim - 1 - ky;
        for (unsigned int kx = 0; kx < filterDim; kx++) {
          int nkx = filterDim - 1 - kx;

          int yy = threadIdx.y + (ky - filterCenter);
          int xx = threadIdx.x + (kx - filterCenter);
          if (xx >= 0 && xx < blockDim.x && yy >=0 && yy < blockDim.y)
            aggregate += in_shared[xx + yy *  blockDim.x] * filter_shared[nky * filterDim + nkx];
        }
      }
      aggregate *= filterFactor;
      if (aggregate > 0) {
        out[x + y * width] = (aggregate > 255) ? 255 : aggregate;
      } else {
        out[x + y * width] = 0;
      }
    }
    // If the pixel needs pixels from other blocks, use global memory 
    else{
      int aggregate = 0;
      for (unsigned int ky = 0; ky < filterDim; ky++) {
        int nky = filterDim - 1 - ky;
        for (unsigned int kx = 0; kx < filterDim; kx++) {
          int nkx = filterDim - 1 - kx;

          int yy = y + (ky - filterCenter);
          int xx = x + (kx - filterCenter);
          if (xx >= 0 && xx < (int) width && yy >=0 && yy < (int) height)
            aggregate += in[xx + yy *  width] * filter_shared[nky * filterDim + nkx];
        }
      }
      aggregate *= filterFactor;
      if (aggregate > 0) {
        out[x + y * width] = (aggregate > 255) ? 255 : aggregate;
      } else {
        out[x + y * width] = 0;
      }
      
    }    
  }
}
/*************************************************************/

/*
 * Get system time to microsecond precision (ostensibly, the same as MPI_Wtime),
 * returns time in seconds
 */
double walltime ( void ) {
  static struct timeval t;
  gettimeofday ( &t, NULL );
  return ( t.tv_sec + 1e-6 * t.tv_usec );
}

void help(char const *exec, char const opt, char const *optarg) {
  FILE *out = stdout;
  if (opt != 0) {
    out = stderr;
    if (optarg) {
      fprintf(out, "Invalid parameter - %c %s\n", opt, optarg);
    } else {
      fprintf(out, "Invalid parameter - %c\n", opt);
    }
  }
  fprintf(out, "%s [options] <input-bmp> <output-bmp>\n", exec);
  fprintf(out, "\n");
  fprintf(out, "Options:\n");
  fprintf(out, "  -i, --iterations <iterations>    number of iterations (1)\n");

  fprintf(out, "\n");
  fprintf(out, "Example: %s in.bmp out.bmp -i 10000\n", exec);
}

int main(int argc, char **argv) {
  /*
    Parameter parsing, don't change this!
   */
  unsigned int iterations = 1;
  char *output = NULL;
  char *input = NULL;
  int ret = 0;

  static struct option const long_options[] =  {
      {"help",       no_argument,       0, 'h'},
      {"iterations", required_argument, 0, 'i'},
      {0, 0, 0, 0}
  };

  static char const * short_options = "hi:";
  {
    char *endptr;
    int c;
    int option_index = 0;
    while ((c = getopt_long(argc, argv, short_options, long_options, &option_index)) != -1) {
      switch (c) {
      case 'h':
        help(argv[0],0, NULL);
        return 0;
      case 'i':
        iterations = strtol(optarg, &endptr, 10);
        if (endptr == optarg) {
          help(argv[0], c, optarg);
          return ERROR_EXIT;
        }
        break;
      default:
        abort();
      }
    }
  }

  if (argc <= (optind+1)) {
    help(argv[0],' ',"Not enough arugments");
    return ERROR_EXIT;
  }
  input = (char *)calloc(strlen(argv[optind]) + 1, sizeof(char));
  strncpy(input, argv[optind], strlen(argv[optind]));
  optind++;

  output = (char *)calloc(strlen(argv[optind]) + 1, sizeof(char));
  strncpy(output, argv[optind], strlen(argv[optind]));
  optind++;

  /*
    End of Parameter parsing!
   */
  
  // Timing variables
  double start;
  double hosttime=0;
  double devicetime=0;

  // CUDA device properties
  hipDeviceProp_t p;
  hipSetDevice(0);
  hipGetDeviceProperties (&p, 0);
  printf("Device compute capability: %d.%d\n", p.major, p.minor);

  
  // Create the BMP image and load it from disk.
  bmpImage *image = newBmpImage(0,0);
  if (image == NULL) {
    fprintf(stderr, "Could not allocate new image!\n");
  }

  if (loadBmpImage(image, input) != 0) {
    fprintf(stderr, "Could not load bmp image '%s'!\n", input);
    freeBmpImage(image);
    return ERROR_EXIT;
  }


  // Create a single color channel image. It is easier to work just with one color
  bmpImageChannel *imageChannel = newBmpImageChannel(image->width, image->height);
  if (imageChannel == NULL) {
    fprintf(stderr, "Could not allocate new image channel!\n");
    freeBmpImage(image);
    return ERROR_EXIT;
  }

  // Create a single color channel image. It is easier to work just with one color (CPU reference)
  bmpImageChannel *referenceImageChannel = newBmpImageChannel(image->width, image->height);
  if (referenceImageChannel == NULL) {
    fprintf(stderr, "Could not allocate new reference image channel!\n");
    freeBmpImage(image);
    return ERROR_EXIT;
  }

  // Extract from the loaded image an average over all colors - nothing else than
  // a black and white representation
  // extractImageChannel and mapImageChannel need the images to be in the exact
  // same dimensions!
  // Other prepared extraction functions are extractRed, extractGreen, extractBlue
  if(extractImageChannel(imageChannel, image, extractAverage) != 0) {
    fprintf(stderr, "Could not extract image channel!\n");
    freeBmpImage(image);
    freeBmpImageChannel(imageChannel);
    return ERROR_EXIT;
  }

  // Extract from the loaded image an average over all colors - nothing else than
  // a black and white representation
  // extractImageChannel and mapImageChannel need the images to be in the exact
  // same dimensions!
  // Other prepared extraction functions are extractRed, extractGreen, extractBlue
  if(extractImageChannel(referenceImageChannel, image, extractAverage) != 0) {
    fprintf(stderr, "Could not extract reference image channel!\n");
    freeBmpImage(image);
    freeBmpImageChannel(referenceImageChannel);
    return ERROR_EXIT;
  }

  // CPU implementation
  bmpImageChannel *processImageChannel = newBmpImageChannel(referenceImageChannel->width, referenceImageChannel->height);
  start=walltime();
  for (unsigned int i = 0; i < iterations; i ++) {
    applyFilter(processImageChannel->data, referenceImageChannel->data, referenceImageChannel->width, referenceImageChannel->height, (int *)laplacian1Filter, 3, laplacian1FilterFactor);
    //Swap the data pointers
    unsigned char ** tmp = processImageChannel->data;
    processImageChannel->data = referenceImageChannel->data;
    referenceImageChannel->data = tmp;
    unsigned char * tmp_raw = processImageChannel->rawdata;
    processImageChannel->rawdata = referenceImageChannel->rawdata;
    referenceImageChannel->rawdata = tmp_raw;
  }
  hosttime+=walltime()-start;  
  freeBmpImageChannel(processImageChannel);

  /******************************* Set up device memory *******************************/
  // Input image
  unsigned char *imageChannelGPU;
  cudaErrorCheck( hipMalloc((void**)&imageChannelGPU, imageChannel->width * imageChannel->height * sizeof(unsigned char)) );
  cudaErrorCheck( hipMemcpy(imageChannelGPU, imageChannel->rawdata, imageChannel->width * imageChannel->height * sizeof(unsigned char), hipMemcpyHostToDevice) );

  // Filter 
  int *filterGPU;
  cudaErrorCheck( hipMalloc((void**)&filterGPU, 3 * 3 * sizeof(int)) );
  cudaErrorCheck( hipMemcpy(filterGPU, laplacian1Filter, 3 * 3 * sizeof(int), hipMemcpyHostToDevice) );
  
  // Output image after each iteration
  unsigned char *processImageChannelGPU;
  cudaErrorCheck( hipMalloc((void**)&processImageChannelGPU, imageChannel->width * imageChannel->height * sizeof(unsigned char)) );
  /************************************************************************************/

  /******************************* Execute GPU Kernel *******************************/
  // GPU implementation
  dim3 gridBlock(ceil(imageChannel->width/BLOCKX), ceil(imageChannel->height/BLOCKY)); //Set the number of blocks accordingly to the image size
  dim3 threadBlock(BLOCKX, BLOCKY); //Each block will have BLOCKX * BLOCKY threads 
  start=walltime();
  for (unsigned int i = 0; i < iterations; i ++) {
    // Call the kernel (Use comments to select between the basic GPU kernel and the shared memory GPU kernel
	
    //Simple GPU kernel
    //device_calculate<<<gridBlock,threadBlock>>>(processImageChannelGPU, imageChannelGPU, imageChannel->width, imageChannel->height, filterGPU, 3, laplacian1FilterFactor);

    // GPU kernel with only input array in shared memory
    //device_shareInput_calculate<<<gridBlock,threadBlock, (BLOCKX * BLOCKY * sizeof(unsigned char))>>>(processImageChannelGPU, imageChannelGPU, imageChannel->width, imageChannel->height, filterGPU, 3, laplacian1FilterFactor);

    // GPU kernel with only filter in shared memory
    //device_shareFilter_calculate<<<gridBlock,threadBlock, (3 * 3 * sizeof(int))>>>(processImageChannelGPU, imageChannelGPU, imageChannel->width, imageChannel->height, filterGPU, 3, laplacian1FilterFactor);
   
    // GPU kernel with both input and filter in GPU memory
    device_shareInputandFilter_calculate<<<gridBlock,threadBlock, (BLOCKX * BLOCKY * sizeof(unsigned char)) + (3 * 3 * sizeof(int))>>>(processImageChannelGPU, imageChannelGPU, imageChannel->width, imageChannel->height, filterGPU, 3, laplacian1FilterFactor);
    
    // Check for errors in the kernel execution
    cudaErrorCheck( hipPeekAtLastError() );
    cudaErrorCheck( hipDeviceSynchronize() );
	
    // Move GPU result to be used as input for the next iteration
    cudaErrorCheck( hipMemcpy(imageChannelGPU, processImageChannelGPU, imageChannel->width * imageChannel->height * sizeof(unsigned char), hipMemcpyDeviceToDevice) );
    
  }
  devicetime+=walltime()-start;
  
  // Get results from the GPU
  cudaErrorCheck( hipMemcpy(imageChannel->rawdata, processImageChannelGPU, imageChannel->width * imageChannel->height * sizeof(unsigned char), hipMemcpyDeviceToHost) );
  /**********************************************************************************/  

  /******************************* Free device memory *******************************/
  // Input image
  cudaErrorCheck( hipFree(imageChannelGPU) );
  
  // Filter
  cudaErrorCheck( hipFree(filterGPU) );

  // Output image
  cudaErrorCheck( hipFree(processImageChannelGPU) );
  /**********************************************************************************/

  // Check if result is correct
  int errors=0;
  
  for(int y=0;y<imageChannel->height;y++) {
    for(int x=0;x<imageChannel->width;x++) {
      int diff=referenceImageChannel->rawdata[x + y * imageChannel->width]-imageChannel->rawdata[x + y * imageChannel->width];
      if(diff<0) diff=-diff;
      if(diff>1) {
        if(errors<10) printf("Error on pixel %d %d: expected %d, found %d\n", x,y,
			     referenceImageChannel->rawdata[x + y * imageChannel->width],
			     imageChannel->rawdata[x + y * imageChannel->width]);
	else if(errors==10) puts("...");
	  errors++;
	}
    }
  }
  if(errors>0) printf("Found %d errors.\n",errors);
  else puts("\nDevice calculations are correct.");

  // Map our single color image back to a normal BMP image with 3 color channels
  // mapEqual puts the color value on all three channels the same way
  // other mapping functions are mapRed, mapGreen, mapBlue
  if (mapImageChannel(image, imageChannel, mapEqual) != 0) {
    fprintf(stderr, "Could not map image channel!\n");
    freeBmpImage(image);
    freeBmpImageChannel(imageChannel);
    return ERROR_EXIT;
  }
  freeBmpImageChannel(imageChannel);
  freeBmpImageChannel(referenceImageChannel);

  // Write the image back to disk
  if (saveBmpImage(image, output) != 0) {
    fprintf(stderr, "Could not save output to '%s'!\n", output);
    freeBmpImage(image);
    return ERROR_EXIT;
  };

  // Print timing results
  printf("\n");
  printf("Host time: %7.3f ms\n",hosttime*1e3);
  printf("Device time: %7.3f ms\n",devicetime*1e3);
  printf("Speedup: %7.3f \n", hosttime/ devicetime);

  ret = 0;
  if (input)
    free(input);
  if (output)
    free(output);
  return ret;
};
© 2020 GitHub, Inc.
Terms
Privacy
Security
Status
Help
Contact GitHub
Pricing
API
Training
Blog
About
