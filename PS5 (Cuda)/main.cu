#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <time.h>

extern "C" {
    #include "libs/bitmap.h"
}

#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__);  }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
       if (code != hipSuccess)
       {
                 fprintf(stderr,"GPUassert: %s %s %s %d\n", hipGetErrorName(code), hipGetErrorString(code), file, line);
                       if (abort) exit(code);
                          
       }
}


// Convolutional Filter Examples, each with dimension 3,
// gaussian filter with dimension 5

int sobelYFilter[] = {-1, -2, -1,
                       0,  0,  0,
                       1,  2,  1};

int sobelXFilter[] = {-1, -0, 1,
                      -2,  0, 2,
                      -1,  0, 1};

int laplacian1Filter[] = { -1,  -4,  -1,
                           -4,  20,  -4,
                           -1,  -4,  -1};

int laplacian2Filter[] = { 0,  1,  0,
                           1, -4,  1,
                           0,  1,  0};

int laplacian3Filter[] = { -1,  -1,  -1,
                           -1,   8,  -1,
                           -1,  -1,  -1};

int gaussianFilter[] = { 1,  4,  6,  4, 1,
                         4, 16, 24, 16, 4,
                         6, 24, 36, 24, 6,
                         4, 16, 24, 16, 4,
                         1,  4,  6,  4, 1 };

const char* filterNames[]       = { "SobelY",     "SobelX",     "Laplacian 1",    "Laplacian 2",    "Laplacian 3",    "Gaussian"     };
int* const filters[]            = { sobelYFilter, sobelXFilter, laplacian1Filter, laplacian2Filter, laplacian3Filter, gaussianFilter };
unsigned int const filterDims[] = { 3,            3,            3,                3,                3,                5              };
float const filterFactors[]     = { 1.0,          1.0,          1.0,              1.0,              1.0,              1.0 / 256.0    };

int const maxFilterIndex = sizeof(filterDims) / sizeof(unsigned int);

void cleanup(char** input, char** output) {
    if (*input)
        free(*input);
    if (*output)
        free(*output);
}

void graceful_exit(char** input, char** output) {
    cleanup(input, output);
    exit(0);
}

void error_exit(char** input, char** output) {
    cleanup(input, output);
    exit(1);
}

// Helper function to swap bmpImageChannel pointers

void swapImageRawdata(pixel **one, pixel **two) {
  pixel *helper = *two;
  *two = *one;
  *one = helper;
}

void swapImage(bmpImage **one, bmpImage **two) {
  bmpImage *helper = *two;
  *two = *one;
  *one = helper;
}

// Apply convolutional filter on image data
void applyFilter(pixel *out, pixel *in, unsigned int width, unsigned int height, int *filter, unsigned int filterDim, float filterFactor) {
  unsigned int const filterCenter = (filterDim / 2);
  for (unsigned int y = 0; y < height; y++) {
    for (unsigned int x = 0; x < width; x++) {
      int ar = 0, ag = 0, ab = 0;
      for (unsigned int ky = 0; ky < filterDim; ky++) {
        int nky = filterDim - 1 - ky;
        for (unsigned int kx = 0; kx < filterDim; kx++) {
          int nkx = filterDim - 1 - kx;

          int yy = y + (ky - filterCenter);
          int xx = x + (kx - filterCenter);
          if (xx >= 0 && xx < (int) width && yy >=0 && yy < (int) height) {
            ar += in[yy*width + xx].r * filter[nky * filterDim + nkx];
            ag += in[yy*width + xx].g * filter[nky * filterDim + nkx];
            ab += in[yy*width + xx].b * filter[nky * filterDim + nkx];
          }
        }
      }

      ar *= filterFactor;
      ag *= filterFactor;
      ab *= filterFactor;
      
      ar = (ar < 0) ? 0 : ar;
      ag = (ag < 0) ? 0 : ag;
      ab = (ab < 0) ? 0 : ab;

      out[y*width +x].r = (ar > 255) ? 255 : ar;
      out[y*width +x].g = (ag > 255) ? 255 : ag;
      out[y*width +x].b = (ab > 255) ? 255 : ab;
    }
  }
}

// Apply cuda filter on image data
__global__ void applyCudaFilter(pixel *out, pixel *in, unsigned int width, unsigned int height, int *filter, unsigned int filterDim, float filterFactor) {
  unsigned int const filterCenter = (filterDim / 2);
  
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  
  // Compute for threads with real pixel value
  if ((x < width) && (y < height)){
    int ar = 0, ag = 0, ab = 0;
    for (unsigned int ky = 0; ky < filterDim; ky++) {
      int nky = filterDim - 1 - ky;
      for (unsigned int kx = 0; kx < filterDim; kx++) {
        int nkx = filterDim - 1 - kx;

        int yy = y + (ky - filterCenter);
        int xx = x + (kx - filterCenter);
        if (xx >= 0 && xx < (int) width && yy >=0 && yy < (int) height) {
          ar += in[yy*width + xx].r * filter[nky * filterDim + nkx];
          ag += in[yy*width + xx].g * filter[nky * filterDim + nkx];
          ab += in[yy*width + xx].b * filter[nky * filterDim + nkx];
        }
      }
    }

    ar *= filterFactor;
    ag *= filterFactor;
    ab *= filterFactor;
    
    ar = (ar < 0) ? 0 : ar;
    ag = (ag < 0) ? 0 : ag;
    ab = (ab < 0) ? 0 : ab;

    out[y*width +x].r = (ar > 255) ? 255 : ar;
    out[y*width +x].g = (ag > 255) ? 255 : ag;
    out[y*width +x].b = (ab > 255) ? 255 : ab;
  }
}

// Apply cuda filter on image data
__global__ void s_applyCudaFilter(pixel *out, pixel *in, unsigned int width, unsigned int height, int *filter, unsigned int filterDim, float filterFactor) {
  unsigned int const filterCenter = (filterDim / 2);
  
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
  
  extern __shared__ pixel sharedArray[];
  
  // Pixels shared memory
  pixel *sharedIn = (pixel*)sharedArray;
  sharedIn[threadIdx.x + threadIdx.y * blockDim.x] = in[x + y * width];

  __syncthreads();

  // Filter shared memory
  int *sharedFilter = (int*)&sharedArray[blockDim.x*blockDim.y];
  
  if (threadIdx.x + threadIdx.y * blockDim.x < filterDim * filterDim){
    sharedFilter[threadIdx.x + threadIdx.y *  blockDim.x] = filter[threadIdx.x + threadIdx.y *  blockDim.x];
  }
  
  __syncthreads();

  // Compute for threads with real pixel value
  if ((x < width) && (y < height)){
    int ar = 0, ag = 0, ab = 0;
    
    // Use shared memory when using pixels from same block
    if (!(threadIdx.x == 0 || threadIdx.x == (blockDim.x-1) || threadIdx.y == 0 || threadIdx.y == (blockDim.y-1))){  
      for (unsigned int ky = 0; ky < filterDim; ky++) {
        int nky = filterDim - 1 - ky;
        for (unsigned int kx = 0; kx < filterDim; kx++) {
          int nkx = filterDim - 1 - kx;

          int yy = threadIdx.y + (ky - filterCenter);
          int xx = threadIdx.x + (kx - filterCenter);
          if (xx >= 0 && xx < blockDim.x && yy >=0 && yy < blockDim.y) {
            ar += sharedIn[yy*blockDim.x + xx].r * sharedFilter[nky * filterDim + nkx];
            ag += sharedIn[yy*blockDim.x + xx].g * sharedFilter[nky * filterDim + nkx];
            ab += sharedIn[yy*blockDim.x + xx].b * sharedFilter[nky * filterDim + nkx];
          }
        }
      }
    
    // Use global memory for pixels outside its own block (but shared filter)
    }else{
      for (unsigned int ky = 0; ky < filterDim; ky++) {
        int nky = filterDim - 1 - ky;
        for (unsigned int kx = 0; kx < filterDim; kx++) {
          int nkx = filterDim - 1 - kx;

          int yy = y + (ky - filterCenter);
          int xx = x + (kx - filterCenter);
          if (xx >= 0 && xx < (int) width && yy >=0 && yy < (int) height) {
            ar += in[yy*width + xx].r * sharedFilter[nky * filterDim + nkx];
            ag += in[yy*width + xx].g * sharedFilter[nky * filterDim + nkx];
            ab += in[yy*width + xx].b * sharedFilter[nky * filterDim + nkx];
          }
        }
      }  
    }
    ar *= filterFactor;
    ag *= filterFactor;
    ab *= filterFactor;
    
    ar = (ar < 0) ? 0 : ar;
    ag = (ag < 0) ? 0 : ag;
    ab = (ab < 0) ? 0 : ab;

    out[y*width +x].r = (ar > 255) ? 255 : ar;
    out[y*width +x].g = (ag > 255) ? 255 : ag;
    out[y*width +x].b = (ab > 255) ? 255 : ab; 
  }
}

void help(char const *exec, char const opt, char const *optarg) {
    FILE *out = stdout;
    if (opt != 0) {
        out = stderr;
        if (optarg) {
            fprintf(out, "Invalid parameter - %c %s\n", opt, optarg);
        } else {
            fprintf(out, "Invalid parameter - %c\n", opt);
        }
    }
    fprintf(out, "%s [options] <input-bmp> <output-bmp>\n", exec);
    fprintf(out, "\n");
    fprintf(out, "Options:\n");
    fprintf(out, "  -k, --filter     <filter>        filter index (0<=x<=%u) (2)\n", maxFilterIndex -1);
    fprintf(out, "  -i, --iterations <iterations>    number of iterations (1)\n");

    fprintf(out, "\n");
    fprintf(out, "Example: %s before.bmp after.bmp -i 10000\n", exec);
}



int main(int argc, char **argv) {
  /*
    Parameter parsing, don't change this!
   */
  unsigned int iterations = 1;
  char *output = NULL;
  char *input = NULL;
  unsigned int filterIndex = 2;

  static struct option const long_options[] =  {
      {"help",       no_argument,       0, 'h'},
      {"filter",     required_argument, 0, 'k'},
      {"iterations", required_argument, 0, 'i'},
      {0, 0, 0, 0}
  };

  static char const * short_options = "hk:i:";
  {
    char *endptr;
    int c;
    int parse;
    int option_index = 0;
    while ((c = getopt_long(argc, argv, short_options, long_options, &option_index)) != -1) {
      switch (c) {
      case 'h':
        help(argv[0],0, NULL);
        graceful_exit(&input,&output);
      case 'k':
        parse = strtol(optarg, &endptr, 10);
        if (endptr == optarg || parse < 0 || parse >= maxFilterIndex) {
          help(argv[0], c, optarg);
          error_exit(&input,&output);
        }
        filterIndex = (unsigned int) parse;
        break;
      case 'i':
        iterations = strtol(optarg, &endptr, 10);
        if (endptr == optarg) {
          help(argv[0], c, optarg);
          error_exit(&input,&output);
        }
        break;
      default:
        abort();
      }
    }
  }

  if (argc <= (optind+1)) {
    help(argv[0],' ',"Not enough arugments");
    error_exit(&input,&output);
  }

  unsigned int arglen = strlen(argv[optind]);
  input = (char*)calloc(arglen + 1, sizeof(char));
  strncpy(input, argv[optind], arglen);
  optind++;

  arglen = strlen(argv[optind]);
  output = (char*)calloc(arglen + 1, sizeof(char));
  strncpy(output, argv[optind], arglen);
  optind++;

  /*
    End of Parameter parsing!
   */


  /*
    Create the BMP image and load it from disk.
   */
  bmpImage *image = newBmpImage(0,0);
  if (image == NULL) {
    fprintf(stderr, "Could not allocate new image!\n");
    error_exit(&input,&output);
  }

  if (loadBmpImage(image, input) != 0) {
    fprintf(stderr, "Could not load bmp image '%s'!\n", input);
    freeBmpImage(image);
    error_exit(&input,&output);
  }

  printf("Apply filter '%s' on image with %u x %u pixels for %u iterations\n", filterNames[filterIndex], image->width, image->height, iterations);


  // implement time measurement from here
  // Start time measurement
  struct timespec start_time, end_time;

  // Here we do the actual computation!
  // image->data is a 2-dimensional array of pixel which is accessed row first ([y][x])
  // image->rawdata is a 1-dimensional array of pixel containing the same data as image->data
  // each pixel is a struct of 3 unsigned char for the red, blue and green colour channel
  bmpImage *processImage = newBmpImage(image->width, image->height);

  // Cuda malloc and memcpy the rawdata from the images, from host side to device side
  int imgXSize = image->width;
  int imgYSize = image->height;
  
  pixel *d_imageRawdata;
  pixel *d_processImageRawdata;
  int *d_filter;

  hipMalloc((void **)&d_imageRawdata, imgXSize*imgYSize*sizeof(pixel));
  hipMalloc((void **)&d_processImageRawdata, imgXSize*imgYSize*sizeof(pixel));
  hipMalloc((void **)&d_filter, filterDims[filterIndex]*filterDims[filterIndex]*sizeof(int));

  hipMemcpy(d_imageRawdata, image->rawdata, imgXSize*imgYSize*sizeof(pixel), hipMemcpyHostToDevice);
  hipMemcpy(d_processImageRawdata, processImage->rawdata, imgXSize*imgYSize*sizeof(pixel), hipMemcpyHostToDevice);
  hipMemcpy(d_filter, filters[filterIndex], filterDims[filterIndex]*filterDims[filterIndex]*sizeof(int), hipMemcpyHostToDevice);

  // Define the gridSize and blockSize, e.g. using dim3 (see Section 2.2. in CUDA Programming Guide)
  int bs;
  int minGridSize;
               
  hipOccupancyMaxPotentialBlockSize(&minGridSize, &bs, s_applyCudaFilter, 0, 0);

  int blockSizeX = floor(bs/32);
  int blockSizeY = floor(bs/32);
  
  //printf("blockSize: %d\n", bs);

  dim3 gridSize(ceil(imgXSize / (float)blockSizeX), ceil(imgYSize / (float)blockSizeY));
  dim3 blockSize(blockSizeX, blockSizeY);

  // Compute shared memory size 
  int sMemSize = blockSizeX*blockSizeY*sizeof(pixel) + filterDims[filterIndex]*filterDims[filterIndex]*sizeof(int);
  
  // Intialize and start CUDA timer
  clock_gettime(CLOCK_MONOTONIC, &start_time);
  for (unsigned int i = 0; i < iterations; i ++) {

    // Parallell implementation 
    s_applyCudaFilter<<<gridSize, blockSize, sMemSize>>>(
                                                        d_processImageRawdata,
                                                        d_imageRawdata,
                                                        image->width,
                                                        image->height,
                                                        d_filter,
                                                        filterDims[filterIndex],
                                                        filterFactors[filterIndex]
    );
    swapImageRawdata(&d_processImageRawdata, &d_imageRawdata);

/*
  // Serial implementation 
  applyFilter(
              processImage->rawdata,
              image->rawdata,
              image->width,
              image->height,
              filters[filterIndex],
              filterDims[filterIndex],
              filterFactors[filterIndex]
  );
  swapImage(&processImage, &image);
*/
  }
  // Check for error
  hipError_t error = hipPeekAtLastError();
  if (error) {
      fprintf(stderr, "A CUDA error has occurred while applying filter: %s\n", hipGetErrorString(error));
  }

  hipDeviceSynchronize();

  // calculate theoretical occupancy
  int maxActiveBlocks;
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, s_applyCudaFilter, bs, sMemSize);


  int device;
  hipDeviceProp_t props;
  hipGetDevice(&device);
  hipGetDeviceProperties(&props, device);

  float occupancy = (maxActiveBlocks * bs / props.warpSize) / 
                    (float)(props.maxThreadsPerMultiProcessor / 
                            props.warpSize);

  printf("Launched blocks of size %d. Theoretical occupancy: %f\n", bs, occupancy);

  //Copy back rawdata from images
  hipMemcpy(image->rawdata, d_imageRawdata, imgXSize*imgYSize*sizeof(pixel), hipMemcpyDeviceToHost);

  //Stop CUDA timer
  // End time measurement and record duration
  clock_gettime(CLOCK_MONOTONIC, &end_time);

  // Calculate and print elapsed time
  float spentTime = ((double) (end_time.tv_sec - start_time.tv_sec)) + ((double) (end_time.tv_nsec - start_time.tv_nsec)) * 1e-9;
  printf("Time spent: %.3f seconds\n", spentTime);

  freeBmpImage(processImage);
  hipFree(d_imageRawdata);
  hipFree(d_processImageRawdata);
  // Write the image back to disk
  if (saveBmpImage(image, output) != 0) {
    fprintf(stderr, "Could not save output to '%s'!\n", output);
    freeBmpImage(image);
    error_exit(&input,&output);
  };

  graceful_exit(&input,&output);
};

